#include "hip/hip_runtime.h"
#include "Mandelbrot.h"

#include <omp.h>

#include "ImageGPU.h"

struct Color {
    uint8_t r, g, b;
    __host__ __device__ constexpr Color() : r(0), g(0), b() {}
    __host__ __device__ constexpr Color(uint8_t r, uint8_t g, uint8_t b) : r(r), g(g), b(b) {}
    __host__ __device__ Color& invert() {
        r = 255 - r;
        g = 255 - g;
        b = 255 - b;
        return *this;
    }
};

constexpr size_t c_paletteSize = 16;

constexpr Color h_palette[c_paletteSize] = {
    Color(66, 30, 15),
    Color(25, 7, 26),
    Color(9, 1, 47),
    Color(4, 4, 73),
    Color(0, 7, 100),
    Color(12, 44, 138),
    Color(24, 82, 177),
    Color(57, 125, 209),
    Color(134, 181, 229),
    Color(211, 236, 248),
    Color(241, 233, 191),
    Color(248, 201, 95),
    Color(255, 170, 0),
    Color(204, 128, 0),
    Color(153, 87, 0),
    Color(106, 52, 3),
};

__constant__ Color d_palette[c_paletteSize];

__device__ __host__ double scale(int x, int rangeSize, double begin, double end) {
    return begin + (end - begin) * x / rangeSize;
}

__device__ __host__ double lerp(double start, double end, double amount) {
    return start + amount * (end - start);
}

__device__ __host__ uint8_t lerp(uint8_t start, uint8_t end, double amount) {
    return start + amount * (static_cast<int>(end) - start);
}

__device__ __host__ Color lerp(Color start, Color end, double amount) {
    return Color(lerp(start.r, end.r, amount), lerp(start.g, end.g, amount), lerp(start.b, end.b, amount));
}

__device__ __host__ Color pickColor(ColorStrategy strategy, int iterations, int maxIterations, int x, int y, const Color palette[c_paletteSize]) {
    switch (strategy) {
        case ColorStrategy::GRAYSCALE: {
            uint8_t color = scale(iterations, maxIterations, 0, 255);
            return Color(color, color, color);
        }
        case ColorStrategy::CONTINUOUS: { 
            // TODO
            double dIterations = static_cast<double>(iterations);
            if (iterations < maxIterations) {
                double logZN = log(static_cast<double>(x * x + y * y)) / 2;
                double nu = log(logZN / log(2.0)) / log(2.0);
                dIterations = dIterations + 1 - nu;
            }
            double fractional = dIterations - floor(static_cast<double>(dIterations));

            Color color1 = palette[static_cast<size_t>(floor(static_cast<double>(dIterations))) % c_paletteSize];
            Color color2 = palette[static_cast<size_t>(floor(static_cast<double>(dIterations)) + 1) % c_paletteSize];

            Color finalColor = lerp(color1, color2, fractional);
            return finalColor;
        }
        case ColorStrategy::ESCAPETIME: {
            Color clr(0, 0, 0);
            if (iterations < maxIterations)
                clr = palette[iterations % c_paletteSize];

            return clr;
        }
    }
    return Color(0, 0, 0);
}

__host__ __device__ int mandelbrotIteration(int pX, int pY, size_t xDim, size_t yDim, int maxIts, double zoom, int xOffset, int yOffset) {
    double scaledXOffset = xOffset / static_cast<int>(xDim) * 3;
    double scaledYOffset = yOffset / static_cast<int>(yDim) * 2;
    double scaledX = scale(pX + xOffset, xDim, -2 / zoom + scaledXOffset, 1 / zoom + scaledXOffset);
    double scaledY = scale(pY + yOffset, yDim, -1 / zoom + scaledYOffset, 1 / zoom + scaledYOffset);

    double x = 0.0;
    double y = 0.0; 
    int it = 0;
    while (x * x + y * y <= 4 && it < maxIts) {
        double tmpX = x * x - y * y + scaledX;
        y = 2 * x * y + scaledY;
        x = tmpX;
    
        ++it;
    }
    return it;
}

template <class ImageType>
__host__ __device__ void colorPixel(ImageType& image, size_t x, size_t y, int maxIts, double zoom, int xOffset, int yOffset, ColorStrategy strategy, bool invertColors, const Color palette[16]) {
    int it = mandelbrotIteration(x, y, image.xDim(), image.yDim(), maxIts, zoom, xOffset, yOffset);

    Color clr = pickColor(strategy, it, maxIts, x, y, palette);
    if (invertColors)
        clr.invert();

    if (strategy == ColorStrategy::GRAYSCALE)
        image(x, y, 0) = clr.r;
    else {
        image(x, y, 0) = clr.r;
        image(x, y, 1) = clr.g;
        image(x, y, 2) = clr.b;
    }
}

Image mandelbrotCPU(size_t size, int maxIts, double zoom, int xOffset, int yOffset, ColorStrategy strategy, bool invertColors) {
    Image image(size * 1.5, size, strategy == ColorStrategy::GRAYSCALE ? 1 : 3);
    size_t xDim = image.xDim();
    size_t yDim = image.yDim();

    #pragma omp parallel for collapse(2)
    for (int x = 0; x < xDim; ++x) {
        for (int y = 0; y < yDim; ++y) {
            colorPixel(image, x, y, maxIts, zoom, xOffset, yOffset, strategy, invertColors, h_palette);
        }
    }

    return image;
}

__device__ int getThreadId() {
    int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    int threadId = threadIdx.y * blockDim.x + threadIdx.x;
    return blockId * (blockDim.x * blockDim.y) + threadId;
}

__global__ void mandelbrotKernel(ImageGPU::Ref image, int maxIts, double zoom, int xOffset, int yOffset, ColorStrategy strategy, bool invertColors) {
    int pixelIndex = getThreadId();
    int y = pixelIndex / image.xDim();
    int x = pixelIndex - y * image.xDim();
    
    if (x < image.xDim() && y < image.yDim())
        colorPixel(image, x, y, maxIts, zoom, xOffset, yOffset, strategy, invertColors, d_palette);
}


Image mandelbrotGPU(size_t size, int maxIts, double zoom, int xOffset, int yOffset, ColorStrategy strategy, bool invertColors) {
    hipMemcpyToSymbol(HIP_SYMBOL(d_palette), h_palette, c_paletteSize * sizeof(Color));
    ImageGPU gpuImage(size * 1.5, size, strategy == ColorStrategy::GRAYSCALE ? 1 : 3);
    size_t xDim = gpuImage.xDim();
    size_t yDim = gpuImage.yDim();

    int suggestedMinGridSize;
    int suggestedBlockSize;
    hipOccupancyMaxPotentialBlockSize(&suggestedMinGridSize, &suggestedBlockSize, mandelbrotKernel);

    size_t blockDimX = sqrt(suggestedBlockSize);
    size_t blockDimY = blockDimX;
    dim3 blockDim(blockDimX, blockDimY);
    dim3 gridDim(ceil(static_cast<double>(xDim) / blockDimX), ceil(static_cast<double>(yDim) / blockDimY));

    mandelbrotKernel<<<gridDim, blockDim>>>(gpuImage.getRef(), maxIts, zoom, xOffset, yOffset, strategy, invertColors);
    hipDeviceSynchronize();

    return gpuImage.toHost();
}