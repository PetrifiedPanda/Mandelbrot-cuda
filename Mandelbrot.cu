#include "hip/hip_runtime.h"
#include "Mandelbrot.h"

#include <omp.h>

#include "ImageGPU.h"

struct Color {
    unsigned char r, g, b;
    __host__ __device__ constexpr Color() : r(0), g(0), b() {}
    __host__ __device__ constexpr Color(unsigned char r, unsigned char g, unsigned char b) : r(r), g(g), b(b) {}
    __host__ __device__ Color& invert() {
        r = 255 - r;
        g = 255 - g;
        b = 255 - b;
        return *this;
    }
};

constexpr size_t c_paletteSize = 16;

constexpr Color h_palette[c_paletteSize] = {
    Color(66, 30, 15),
    Color(25, 7, 26),
    Color(9, 1, 47),
    Color(4, 4, 73),
    Color(0, 7, 100),
    Color(12, 44, 138),
    Color(24, 82, 177),
    Color(57, 125, 209),
    Color(134, 181, 229),
    Color(211, 236, 248),
    Color(241, 233, 191),
    Color(248, 201, 95),
    Color(255, 170, 0),
    Color(204, 128, 0),
    Color(153, 87, 0),
    Color(106, 52, 3),
};

__constant__ Color d_palette[c_paletteSize] = {
    Color(66, 30, 15),
    Color(25, 7, 26),
    Color(9, 1, 47),
    Color(4, 4, 73),
    Color(0, 7, 100),
    Color(12, 44, 138),
    Color(24, 82, 177),
    Color(57, 125, 209),
    Color(134, 181, 229),
    Color(211, 236, 248),
    Color(241, 233, 191),
    Color(248, 201, 95),
    Color(255, 170, 0),
    Color(204, 128, 0),
    Color(153, 87, 0),
    Color(106, 52, 3),
};

__device__ __host__ double scale(int x, int rangeSize, double begin, double end) {
    return begin + (end - begin) * x / rangeSize;
}

__device__ __host__ double lerp(double start, double end, double amount) {
    return start + amount * (end - start);
}

__device__ __host__ Color pickColor(ColorStrategy strategy, int iterations, int maxIterations, int x, int y, const Color palette[c_paletteSize]) {
    switch (strategy) {
        case ColorStrategy::GRAYSCALE: {
            unsigned char color = scale(iterations, maxIterations, 0, 255);
            return Color(color, color, color);
        }
        case ColorStrategy::CONTINUOUS: { 
            // TODO
            double dIterations = iterations;
            if (iterations < maxIterations) {
                double logZN = log(static_cast<double>(x * x + y * y)) / 2;
                double nu = log(logZN / log(2.0)) / log(2.0);
                dIterations = dIterations + 1 - nu;
            }
            double fractional = dIterations - floor(static_cast<double>(dIterations));

            Color color1 = palette[static_cast<size_t>(floor(static_cast<double>(dIterations))) % c_paletteSize];
            Color color2 = palette[static_cast<size_t>(floor(static_cast<double>(dIterations)) + 1) % c_paletteSize];
        
            return Color(lerp(color1.r, color2.r, fractional), lerp(color1.g, color2.g, fractional), lerp(color1.b, color2.b, fractional));
        }
        case ColorStrategy::ESCAPETIME: {
            Color clr(0, 0, 0);
            if (iterations < maxIterations)
                clr = palette[iterations % c_paletteSize];

            return clr;
        }
    }
    return Color(0, 0, 0);
}

__host__ __device__ int mandelbrotIteration(int pX, int pY, size_t rows, size_t cols, int maxIts) {
    double scaledX = scale(pX, rows, -2, 1);
    double scaledY = scale(pY, cols, -1, 1);

    double x = 0.0;
    double y = 0.0; 
    int it = 0;
    while (x * x + y * y <= 4 && it < maxIts) {
        double tmpX = x * x - y * y + scaledX;
        y = 2 * x * y + scaledY;
        x = tmpX;
    
        ++it;
    }
    return it;
}

template <class ImageType>
__host__ __device__ void colorPixel(ImageType& image, size_t x, size_t y, int maxIts, ColorStrategy strategy, bool invertColors, const Color palette[16]) {
    int it = mandelbrotIteration(x, y, image.rows(), image.cols(), maxIts);

    Color clr = pickColor(strategy, it, maxIts, x, y, palette);
    if (invertColors)
        clr.invert();

    if (strategy == ColorStrategy::GRAYSCALE)
        image(x, y, 0) = clr.r;
    else {
        image(x, y, 0) = clr.r;
        image(x, y, 1) = clr.g;
        image(x, y, 2) = clr.b;
    }
}

Image mandelbrotCPU(size_t size, int maxIts, ColorStrategy strategy, bool invertColors) {
    Image image(size * 1.5, size, strategy == ColorStrategy::GRAYSCALE ? 1 : 3);
    size_t rows = image.rows();
    size_t cols = image.cols();

    #pragma omp parallel for collapse(2)
    for (int x = 0; x < rows; ++x) {
        for (int y = 0; y < cols; ++y) {
            colorPixel(image, x, y, maxIts, strategy, invertColors, h_palette);
        }
    }

    return image;
}

__device__ int getThreadId() {
    int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    int threadId = threadIdx.y * blockDim.x + threadIdx.x;
    return blockId * (blockDim.x * blockDim.y) + threadId;
}

__global__ void mandelbrotKernel(ImageGPU::Ref image, int maxIts, ColorStrategy strategy, bool invertColors) {
    int pixelIndex = getThreadId();
    int x = pixelIndex / image.cols();
    int y = pixelIndex - x * image.cols();
    
    if (x < image.rows() && y < image.cols())
        colorPixel(image, x, y, maxIts, strategy, invertColors, d_palette);
}


Image mandelbrotGPU(size_t size, int maxIts, ColorStrategy strategy, bool invertColors) {
    ImageGPU gpuImage(size * 1.5, size, strategy == ColorStrategy::GRAYSCALE ? 1 : 3);
    size_t rows = gpuImage.rows();
    size_t cols = gpuImage.cols();

    int suggestedMinGridSize;
    int suggestedBlockSize;
    hipOccupancyMaxPotentialBlockSize(&suggestedMinGridSize, &suggestedBlockSize, mandelbrotKernel);

    size_t blockDimX = sqrt(suggestedBlockSize);
    size_t blockDimY = blockDimX;
    dim3 blockDim(blockDimX, blockDimY);
    dim3 gridDim(ceil(static_cast<double>(cols) / blockDimX), ceil(static_cast<double>(rows) / blockDimY));

    mandelbrotKernel<<<gridDim, blockDim>>>(gpuImage.getRef(), maxIts, strategy, invertColors);
    hipDeviceSynchronize();

    return gpuImage.toHost();
}